#include <CudaHelper.cuh>
#include <iostream>

void checkCudaError(hipError_t err, const std::string &msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}