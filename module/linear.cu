#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <memory>
#include "cudakernels.cuh"
#include "linear.cuh"
#include "tensor.cuh"
#include "tensorops.cuh"

void xavier_init(Tensor *t, int fan_in, int fan_out) {
    float limit = std::sqrt(6.0f / (fan_in + fan_out));
    for (int i = 0; i < t->batch_size * t->features; ++i) {
        float r = static_cast<float>(std::rand()) / RAND_MAX;  // [0,1]
        r = r * 2.0f * limit - limit;  // [ -limit, +limit ]
        t->data[i] = r;
    }
}

Linear::Linear(int in_f, int out_f) {
    in_features = in_f;
    out_features = out_f;

    weight = tensor_create(in_f, out_f, 1);  // (K, N)
    xavier_init(this->weight, in_features, out_features);        // 新增 Xavier 初始化
    bias = tensor_create(1, out_f, 1);  // (1, N)
}

Tensor *Linear::forward(Tensor *input) {
    Tensor *out = tensor_matmul(input, this->weight);  // z = x × y
    // out = tensor_add_bias(out, this->bias); // z = x × y + b
    return out;
}

Tensor *Linear::_tensor() { return this->weight; }

Linear::~Linear() {
    tensor_free(this->weight);
    tensor_free(this->bias);
}

void Linear::print_weight(const std::string &name) const {
    if (name.empty()) {
        std::cout << "Weight: " << std::endl;
    } else {
        std::cout << name << ": " << std::endl;
    }
    tensor_print(this->weight);
}

void Linear::print_grad(const std::string &name) const {
    if (name.empty()) {
        std::cout << "Grad: " << std::endl;
    } else {
        std::cout << name << ": " << std::endl;
    }
    tensor_print_grad(this->weight);
}