#include "activationkernels.cuh"
#include "cudakernels.cuh"
#include "tensorops.cuh"

#include <hip/hip_runtime.h>

Tensor *tensor_matmul(Tensor *a, Tensor *b) {
    int M = a->batch_size;
    int K = a->features;
    int N = b->features;

    if (K != b->batch_size) {
        printf("Shape mismatch in tensor_matmul: (%d x %d) × (%d x %d)\n", M, K, b->batch_size, N);
        return NULL;
    }

    Tensor *out = tensor_create(M, N, a->requires_grad || b->requires_grad);

    // Device memory
    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, M * K * sizeof(float));
    hipMalloc(&d_b, K * N * sizeof(float));
    hipMalloc(&d_out, M * N * sizeof(float));

    hipMemcpy(d_a, a->data, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b->data, K * N * sizeof(float), hipMemcpyHostToDevice);

    // Kernel launch
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    matrixMultiplyKernel<<<gridSize, blockSize>>>(d_a, d_b, d_out, M, K, N);
    hipDeviceSynchronize();  // 🔥 必須加

    // Copy result back
    hipMemcpy(out->data, d_out, M * N * sizeof(float), hipMemcpyDeviceToHost);
    // for (int i = 0; i < M * N; i++) {
    //     printf("dout[%d] = %f\n", i, out->data[i]);
    // }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // AutoGrad
    if (a->requires_grad || b->requires_grad) {
        tensor_add_dependency(out, a, tensor_matmul_backward_a);
        tensor_add_dependency(out, b, tensor_matmul_backward_b);
    }

    return out;
}

Tensor *tensor_matmul_backward_a(Tensor *a, Tensor *b, Tensor *grad_out) {
    int M = a->batch_size;
    int K = a->features;
    int N = b->features;

    Tensor *grad_a = tensor_create(M, K, 0);  // 新建 Tensor 儲存梯度

    // GPU 記憶體配置
    float *d_grad_out, *d_b, *d_grad_a;
    hipMalloc(&d_grad_out, M * N * sizeof(float));
    hipMalloc(&d_b, K * N * sizeof(float));
    hipMalloc(&d_grad_a, M * K * sizeof(float));

    hipMemcpy(d_grad_out, grad_out->data, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b->data, K * N * sizeof(float), hipMemcpyHostToDevice);

    // GPU: 轉置 b 並相乘
    float *d_b_T;
    hipMalloc(&d_b_T, N * K * sizeof(float));
    dim3 blockDim(16, 16);
    dim3 gridDim((K + 15) / 16, (N + 15) / 16);
    matrixTransposeKernel<<<gridDim, blockDim>>>(d_b, d_b_T, K, N);
    hipDeviceSynchronize();

    gridDim = dim3((K + 15) / 16, (M + 15) / 16);
    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_grad_out, d_b_T, d_grad_a, M, N, K);
    hipDeviceSynchronize();

    hipMemcpy(grad_a->data, d_grad_a, M * K * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_grad_out);
    hipFree(d_b);
    hipFree(d_b_T);
    hipFree(d_grad_a);

    return grad_a;
}

Tensor *tensor_matmul_backward_b(Tensor *a, Tensor *b, Tensor *grad_out) {
    int M = a->batch_size;
    int K = a->features;
    int N = b->features;

    Tensor *grad_b = tensor_create(K, N, 0);  // 新建 Tensor 儲存梯度

    float *d_a, *d_a_T, *d_grad_out, *d_grad_b;
    hipMalloc(&d_a, M * K * sizeof(float));
    hipMalloc(&d_a_T, K * M * sizeof(float));
    hipMalloc(&d_grad_out, M * N * sizeof(float));
    hipMalloc(&d_grad_b, K * N * sizeof(float));

    hipMemcpy(d_a, a->data, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_grad_out, grad_out->data, M * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((M + 15) / 16, (K + 15) / 16);
    matrixTransposeKernel<<<gridDim, blockDim>>>(d_a, d_a_T, M, K);
    hipDeviceSynchronize();

    gridDim = dim3((N + 15) / 16, (K + 15) / 16);
    matrixMultiplyKernel<<<gridDim, blockDim>>>(d_a_T, d_grad_out, d_grad_b, K, M, N);
    hipDeviceSynchronize();

    hipMemcpy(grad_b->data, d_grad_b, K * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_a_T);
    hipFree(d_grad_out);
    hipFree(d_grad_b);

    return grad_b;
}

void tensor_backward(Tensor *self, Tensor *grad_out) {
    if (!self->requires_grad) return;

    if (self->num_deps == 2) {
        Dependency *dep0 = &self->deps[0];
        Dependency *dep1 = &self->deps[1];
        if (dep0->backward_fn) {
            Tensor *grad_a = dep0->backward_fn(dep0->tensor, dep1->tensor, grad_out);
            tensor_grad(dep0->tensor, grad_a);
            tensor_backward(dep0->tensor, grad_a);
            tensor_free(grad_a);
        }
        if (dep1 && dep1->backward_fn) {
            Tensor *grad_b = dep1->backward_fn(dep0->tensor, dep1->tensor, grad_out);
            tensor_grad(dep1->tensor, grad_b);
            tensor_free(grad_b);
        }
    } else if (self->num_deps == 1) {
        Dependency *dep = &self->deps[0];
        if (dep->backward_fn) {
            Tensor *grad = dep->backward_fn(dep->tensor, NULL, grad_out);
            tensor_grad(dep->tensor, grad);
            tensor_backward(dep->tensor, grad);
            tensor_free(grad);
        }
    } else {
        printf("No dependencies found for backward pass.\n");
    }
}

void tensor_print_graph_dot_rec(Tensor *self) {
    if (!self) return;

    printf("  \"%p\" [label=\"Tensor %p\"];\n", self, self);
    for (int i = 0; i < self->num_deps; ++i) {
        Dependency *dep = &self->deps[i];
        if (dep && dep->tensor) {
            printf("  \"%p\" -> \"%p\";\n", self, dep->tensor);
            tensor_print_graph_dot_rec(dep->tensor);
        }
    }
}

void tensor_print_graph_dot(Tensor *self) {
    printf("digraph G {\n");
    tensor_print_graph_dot_rec(self);
    printf("}\n");
}

void tensor_grad(Tensor *t, Tensor *grad) {
    if (!t->requires_grad) {
        printf("Tensor does not require gradient.\n");
        return;
    }
    if (!t->grad) {
        t->grad = (float *)calloc(t->batch_size * t->features, sizeof(float));
    }
    for (int i = 0; i < t->batch_size * t->features; ++i) {
        t->grad[i] = -grad->data[i];
    }
}

Tensor *tensor_relu_backward(Tensor *x, Tensor *n, Tensor *grad_out) {
    int size = x->batch_size * x->features;

    Tensor *grad_x = tensor_create(x->batch_size, x->features, 0);

    float *d_x, *d_grad_out, *d_grad_x;
    hipMalloc(&d_x, size * sizeof(float));
    hipMalloc(&d_grad_out, size * sizeof(float));
    hipMalloc(&d_grad_x, size * sizeof(float));

    hipMemcpy(d_x, x->data, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_grad_out, grad_out->data, size * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    reluBackwardKernel<<<gridSize, blockSize>>>(d_x, d_grad_out, d_grad_x, size);
    hipDeviceSynchronize();

    hipMemcpy(grad_x->data, d_grad_x, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_grad_out);
    hipFree(d_grad_x);

    return grad_x;
}

Tensor *tensor_relu(Tensor *x) {
    Tensor *out = tensor_create(x->batch_size, x->features, x->requires_grad);
    int size = x->batch_size * x->features;

    float *d_x, *d_out;
    hipMalloc(&d_x, size * sizeof(float));
    hipMalloc(&d_out, size * sizeof(float));

    hipMemcpy(d_x, x->data, size * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    reluKernel<<<gridSize, blockSize>>>(d_x, d_out, size);
    hipMemcpy(out->data, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_out);

    // 建立依賴鏈
    if (x->requires_grad) {
        tensor_add_dependency(out, x, tensor_relu_backward);
    }

    return out;
}

void tensor_print_grad(Tensor *t) {
    if (!t) {
        printf("Tensor is NULL\n");
        return;
    }
    printf("Tensor grad: ");
    for (int i = 0; i < t->batch_size * t->features; ++i) {
        printf("%f ", t->grad[i]);
    }
    printf("\n");
}

void fill_tensor_with_random(Tensor *t) {
    FILE *fp = fopen("/dev/urandom", "rb");
    if (!fp) {
        perror("fopen");
        exit(1);
    }
    for (int i = 0; i < t->batch_size * t->features; i++) {
        u_int32_t rand_int;
        fread(&rand_int, sizeof(rand_int), 1, fp);
        t->data[i] = 2.0f * (rand_int / (double)UINT32_MAX) - 1.0f;
    }
    fclose(fp);
}

Tensor *tensor_clone(Tensor *t) {
    Tensor *clone = (Tensor *)malloc(sizeof(Tensor));
    clone->batch_size = t->batch_size;
    clone->features = t->features;
    clone->requires_grad = t->requires_grad;
    clone->data = (float *)malloc(t->batch_size * t->features * sizeof(float));
    clone->grad = t->requires_grad ? (float *)malloc(t->batch_size * t->features * sizeof(float)) : NULL;
    memcpy(clone->data, t->data, t->batch_size * t->features * sizeof(float));
    return clone;
}